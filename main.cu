#include "hip/hip_runtime.h"

#define THREADS_PER_BLOCK 1024
#include <stdio.h>
#include <vector>
#include <assert.h>
#include <string>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

#include <GLFW/glfw3.h>
#include "imgui/imgui.h"
#include "imgui/backends/imgui_impl_glfw.h"
#include "imgui/backends/imgui_impl_opengl3.h"
#include <iostream>
//------------------------------------------------Predifining Functions
std::vector<uint16_t> ReadingFiles(char* fileName, int& height, int& width);
void outPutFile(std::vector<uint8_t> pixelData, int height, int width);
void Loop(std::vector<uint8_t> pixelData, int height, int width);
//------------------------------------------------Color operations
__device__
void Color(uint16_t number, uint8_t& n)
{
	// this is the solution if little indian
	uint8_t first_8_bits = number & 0b11111111; first_8_bits = first_8_bits >> 4;
	number = number >> 8;
	n = number & 0b11111111; n = n >> 4;
	// so now we have smth like bit1 -> 10101011, bit0 -> 10101011;
	n = n & 0b1111; // basicly the paddings are throun away
	// now we have 2 4 bit numbers and when combining them OR || XOR
	n = n << 4;
	n |= first_8_bits;
	// the second number is our putput
}

__global__ void Checker(uint16_t* d_Data, uint8_t* cpy_Data, int width, int height)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y; 
	if (x < width && x >= 0 
		&& y < height && y>=0)
	{
		int calc = y * width + x;  //their scope is threadLifeTime
		uint8_t n = 0;
		Color(d_Data[calc], n);
		//h   !w
		short idx = (y & 1) + !(x & 1);
		cpy_Data[3 * calc + 0] = 0;//r
		cpy_Data[3 * calc + 1] = 0;//g
		cpy_Data[3 * calc + 2] = 0;//b
		cpy_Data[3 * calc + idx] = n;
	}
}
//TODO::clean up the code, imo create a class for opengl and split the GetCudaRdy to more functions
__host__
void GetCudaRdy(std::vector<uint8_t> &h_cpy, const std::vector<uint16_t>& data, const int& height, const int& width)
{
	int size = height * width;
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	uint16_t* d_data;
	uint8_t* cpyData;

	hipStream_t stream;
	hipError_t cudaStatus = hipError_t(0);
	cudaStatus = hipSetDevice(0);
	assert(cudaStatus == hipSuccess, "you do not have cuda capable device!");
	cudaStatus = hipStreamCreate(&stream);

	cudaStatus = hipMalloc((void**)&d_data, data.size() * sizeof(uint16_t));//sizeof(uint16_t) * size
	assert(cudaStatus == hipSuccess, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&cpyData, sizeof(uint8_t) * h_cpy.size() * 3);
	assert(cudaStatus == hipSuccess, "hipMalloc failed!");
	// learn to get stram and make the function async
	auto start = std::chrono::high_resolution_clock::now();
	
	cudaStatus = hipMemcpyAsync(d_data, data.data(), sizeof(uint16_t) * size, hipMemcpyHostToDevice, stream);
	assert(cudaStatus == hipSuccess, "not able to tansfer Data!");

	cudaStatus = hipMemcpyAsync(cpyData, h_cpy.data(), sizeof(uint8_t) * size * 3, hipMemcpyHostToDevice, stream);
	assert(cudaStatus == hipSuccess, "not able to tansfer Data!");// here i am actually not in need to transfer data, but i wanted to see if it makes difference
	dim3 sizeOfBlock(((width + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), height); // 4 , 2

	Checker << <sizeOfBlock, THREADS_PER_BLOCK, 0, stream >> > (d_data, cpyData, width, height);

	cudaStatus = hipMemcpyAsync(h_cpy.data(), cpyData, sizeof(uint8_t) * size * 3, hipMemcpyDeviceToHost, stream);
	cudaStatus = hipStreamSynchronize(stream);
	//cudaStatus = hipDeviceSynchronize();
	
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
	printf("%d -> is the time measured", duration);

	hipFree(cpyData);
	hipFree(d_data);// it was said to -> hipFree ( void* devPtr )Frees memory on the device.
	hipStreamDestroy(stream);
}

bool IfFileCanOpen(const char* fileName, const std::string width, const std::string height) {
	FILE* file = fopen(fileName, "r");
	if (file == 0)
	{
		printf("not able to open the file for reading!\n");
		return false;
	}
	fclose(file);
	for (int i = 0; i < fmax(width.size(), height.size()); i++)
	{
		if (width[i] > '9' || width[i] < '0'
			|| height[i] > '9' || height[i] < '0')
		{
			printf("you have declared invalid numbers for width and height\n");
			return false;
		}
	}
	return true;
}
void guiClearColor()
{
	glfwPollEvents();
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT);
}
int main()
{
	char* fileName = "fileToRead.raw";
	int width = 3840, height = 1920;
	const std::vector<uint16_t> &data = ReadingFiles(fileName, height, width);// reading the 

	std::vector<uint8_t> h_cpy;
	h_cpy.resize(width * height * 3);

	GetCudaRdy(h_cpy, data, height, width);
	outPutFile(h_cpy, height, width);// ofstreaming the files
	return 0;
}

std::vector<uint16_t> ReadingFiles(char* fileName, int& height, int& width)
{
	FILE* rdFile = fopen(fileName, "rb+");
	std::vector<uint16_t> data;
	if (rdFile == 0) {
		printf("no file found!");
		return data; 
	}
	data.resize(height * width);
	fread(reinterpret_cast<char*>(&data[0]), 2, height * width, rdFile);
	fclose(rdFile);
	return data;
}

void Loop(std::vector<uint8_t> pixelData, int height, int width)
{
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);

	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	if (!glfwInit()) {
		throw "glfwInit() FAILED!";
	}

	GLFWwindow* window = glfwCreateWindow(640, 480, "My Title", NULL, NULL);

	if (!window) {
		glfwTerminate();
		throw "no window created";
	}

	glfwMakeContextCurrent(window);
	glfwSwapInterval(1);

	IMGUI_CHECKVERSION();
	ImGui::CreateContext();

	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init("#version 330");

	bool is_show = true;
	bool oneImage = false;
	bool twoImages = false;
	GLuint texture;
	while (!glfwWindowShouldClose(window))
	{
		glfwPollEvents();
		glClearColor(0.45f, 0.55f, 0.60f, 1.00f);
		glClear(GL_COLOR_BUFFER_BIT);

		// feed inputs to dear imgui, start new frame
		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();

		// rendering our geometries

		// render your GUI
		ImGui::Begin("Demo window");
		ImGui::Button("Hello!");
		ImGui::End();

		// Render dear imgui into screen
		ImGui::Render();
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

		int display_w, display_h;
		glfwGetFramebufferSize(window, &display_w, &display_h);
		glViewport(0, 0, display_w, display_h);
		glfwSwapBuffers(window);
	}

	ImGui_ImplGlfw_Shutdown();
	ImGui_ImplOpenGL3_Shutdown();
	ImGui::DestroyContext();
	glfwTerminate();

	glfwDestroyWindow(window);
}
void outPutFile(std::vector<uint8_t> pixelData, int height, int width)
{
	FILE* fileWr;
	fileWr = fopen("writingFile.ppm", "w+");
	fprintf(fileWr, "%s %d %d %d ", "P6", width, height, 255);
	fclose(fileWr);

	fileWr = fopen("writingFile.ppm", "ab+");
	fwrite(reinterpret_cast<const char*>(&pixelData[0]), 1, sizeof(uint8_t) * width * height * 3, fileWr);
	fclose(fileWr);
	fileWr = nullptr;
}