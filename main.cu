#include "hip/hip_runtime.h"
#include <GLFW/glfw3.h>
#include "imgui/imgui.h"
#include "imgui/backends/imgui_impl_glfw.h"
#include "imgui/backends/imgui_impl_opengl3.h"

#include <stdio.h>
#include <vector>
#include <string>
#include <chrono>

#include <stdio.h>
#include <stdlib.h>
#define STB_IMAGE_IMPLEMENTATION
#include "../../header/stb_image.h"
//#define STB_IMAGE_RESIZE_IMPLEMENTATION
//#include "build/stb_image_resize.h"
#include "../../header/CudaClass.h"
#include <iostream>
#include <inttypes.h>

//extern std::vector<uint8_t> result;
extern std::vector<uint16_t> matcher_result;

bool load(std::vector<uint8_t> &image,  int &width, int &height, int& channels)
{
	unsigned char* img = stbi_load("image.jpg", &width, &height, &channels, 1);
	//stbir_resize_uint8(img, width, height, 0, img, width, height, 0, 1);
	if (img == NULL) { return false; }
	image.resize(width * height);
	memcpy(&image[0], img, image.size());
	if (image.empty()){	return false; }
	stbi_image_free(img);
	return true;
}

std::vector<uint16_t> ReadingFiles(char* fileName, int height, int width);

void bindTexture(GLuint texture)
{
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glPixelStorei(GL_UNPACK_ROW_LENGTH, 0);
}
void onNewFrame()
{
	glfwPollEvents();
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT);

	ImGui_ImplOpenGL3_NewFrame();
	ImGui_ImplGlfw_NewFrame();
	ImGui::NewFrame();
}
void createContext(GLFWwindow* &window)
{
	glfwMakeContextCurrent(window);
	glfwSwapInterval(0);

	IMGUI_CHECKVERSION();
	ImGui::CreateContext();

	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init("#version 330");
}

void Loop(const std::vector<uint16_t>& data, const int height, const int width)
{
	std::vector<uint8_t> h_result;
	int size = height * width * 3;
	h_result.resize(size);

	Cuda cuda(h_result, data, height, width);
	cuda.startup(size);

	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);

	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	if (!glfwInit()) {
		throw "glfwInit() FAILED!";
	}

	GLFWwindow* window = glfwCreateWindow(800, 600, "Raw-File Viewer", NULL, NULL);

	if (!window) {
		glfwTerminate();
		throw "no window created";
	}

	cuda.sync();
	createContext(window);

	bool is_show = true;
	GLuint texture;
	glGenTextures(1, &texture);

	while (!glfwWindowShouldClose(window))
	{
		cuda.rawValue();
		onNewFrame();
		ImGui::Begin("raw Image", &is_show);
		bindTexture(texture);
		cuda.sync();
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, width, height, 0, GL_RGB, GL_UNSIGNED_BYTE, h_result.data());
		ImGui::Image(reinterpret_cast<void*>(static_cast<intptr_t>(texture)), ImVec2(800, 600));
		ImGui::End();
		ImGui::Text("Application average %.3f ms/frame (%.1f FPS)", 1000.0f / ImGui::GetIO().Framerate, ImGui::GetIO().Framerate);
		ImGui::Render();
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
		glDeleteTextures(sizeof(texture), &texture);
		glfwSwapBuffers(window);
	}

	cuda.outPutFile();
	ImGui_ImplGlfw_Shutdown();
	ImGui_ImplOpenGL3_Shutdown();
	ImGui::DestroyContext();
	glfwTerminate();
	glfwDestroyWindow(window);
}

struct NVProf {
	NVProf(const char* name) {
		nvtxRangePush(name);
	}
	~NVProf() {
		nvtxRangePop();
	}
};

#define NVPROF_SCOPE(X) NVProf __nvprof(X);

void MatchKernel_Result(const std::vector<uint8_t>& data, const int height, const int width)
{
	auto time = std::chrono::duration<double>();

	hipStream_t stream;
	hipStreamCreate(&stream);
	
	std::vector<uint16_t> h_result;
	int size = height * width;
	float a = 5.0f;
	
	std::vector<descriptor_t> query;
	std::vector<descriptor_t> train;
	CudaKeypoints cuda(data, height, width);

	/*function to get the smallest elements from an array
	 getSmallElements()*/
	std::vector<float2> input(1000);
	std::vector<float2> output(1000);
	input[0].x = 1000.f;
	input[0].y = 0;
	for (int i = 1; i < 1000; ++i)
	{
		input[i].x = (float(rand()) / float((RAND_MAX)) * a);
		input[i].y = (float(rand()) / float((RAND_MAX)) * a);
	}
	cuda.getSmallElements(input, output, 5.0f, stream);
	for (int i = 0; i < output.size(); ++i)
	{
		if (output[i].y > 5.0f)
		{
			printf("there is an impostor in the group => %f\n", output[i].y);
		}
	}
	 /*~getSmallElements()

	keypoitns for query and train
	 Kernel()*/
	cuda.startup(size, leftPoint.size() / 2);

	{
		NVPROF_SCOPE("Kernel plus copy from cpu-gpu");
		cuda.cudaUploadKeypoints(leftPoint);
		cuda.Kernel(leftPoint.size() / 2);
		cuda.cudaMemcpyD2H(query, leftPoint.size() / 2);
	}
	cuda.sync(stream);

	{
		NVPROF_SCOPE("Second kernel cpu-gpu");
		cuda.cudaUploadKeypoints(rightPoint);
		cuda.Kernel(rightPoint.size() / 2);
		cuda.cudaMemcpyD2H(train, rightPoint.size() / 2);
	}
	cuda.sync(stream);
	/* ~Kernel

	 meatcher*/
	cuda.MemoryAllocationAsync(stream, query.size(), train.size());
	
	{	
		for (int i = 0; i < 1000; ++i)
		{
			NVPROF_SCOPE("for a single iteration on match kernel");
			cuda.MemcpyUploadAsyncForMatches(stream, query, train);
			cuda.match_gpu_caller(stream, query.size(), train.size());
			cuda.downloadAsync(stream, h_result, query.size());
			cuda.sync(stream);
		}
	}
	time = std::chrono::microseconds::zero();
	cuda.cudaFreeAcyncMatcher(stream);
	//for (int i = 0; i < 100; ++i)
	//{
	//	if (h_result[i] != matcher_result[i])
	//	{
	//		printf("not equal\n");
	//	}
	//}

	//~meatcher

	/* memory allcation managed
	 if there ain't memMallocManaged in d.cuda declare memory using the __managed__ keyword*/
	//cuda.MemoryAllocationManagedForMatches(query.size(), train.size());
	//{
	//	auto start = std::chrono::high_resolution_clock::now();
	//	NVPROF_SCOPE("managedAllocationPipeline");

	//		cuda.AttachMemAsync(stream, query, train);
	//		cuda.match_gpu_caller(stream, query.size(), train.size());
	//		cuda.sync(stream);
	//	auto end = std::chrono::high_resolution_clock::now();
	//	time = end - start;
	//}
	//printf("%f\n", time.count());
	// //~memory allcation managed
	//cuda.cudaFreeManaged();
}
void RawFileConverter()
{
	char* fileName = "fileToRead.raw";
	int width = 3840, height = 1920;
	const std::vector<uint16_t> &data = ReadingFiles(fileName, height, width);
	Loop(data, height, width);
}

void MatchKernel()
{
	int width = 1920, height = 1200;
	int channels;
	std::vector<uint8_t> data;
	if (!load(data, height, width, channels)) { throw "cannot load an image"; }
	MatchKernel_Result(data, height, width);
}
int main()
{
	//RawFileConverter();
	//KeypointTest();
	MatchKernel();
	return 0;
}

std::vector<uint16_t> ReadingFiles(char* fileName, int height, int width)
{
	FILE* rdFile = fopen(fileName, "rb+");
	std::vector<uint16_t> data;
	if (rdFile == 0) {
		printf("no file found!");
		return data; 
	}
	int size = height * width;
	data.resize(size);
	fread(reinterpret_cast<char*>(&data[0]), 2, size, rdFile);
	fclose(rdFile);
	return data;
}
