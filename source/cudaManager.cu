#include "cudaManager.h"
#include "nvtx3/nvToolsExt.h"

#include <chrono>
#include <cassert>

NVProf::NVProf(const char* name) {
    nvtxRangePush(name);
}

NVProf::~NVProf() {
    nvtxRangePop();
}
#define NVPROF_SCOPE(X) NVProf __nvprof(X);
ImageParams::ImageParams(
    const int32_t _height,
    const int32_t _width,
    const int32_t _stride,
    const int32_t _bpp
) : height(_height)
, width(_width)
, stride(_stride)
, bpp(_bpp) {}

Cuda::Cuda(
    ImageParams& _params
) :params(_params)
, d_data(nullptr)
, d_result(nullptr) {
    cudaStatus = hipError_t(0);
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	cudaStatus = hipSetDevice(0);
	assert(cudaStatus == hipSuccess && "you do not have cuda capable device!");
	cudaStatus = hipStreamCreate(&stream);
}

__host__
void Cuda::memoryAllocation(hipStream_t providedStream, const size_t sizeInBytes, const size_t resultSize) {
    hipStream_t useStream = providedStream == nullptr ? stream : providedStream;
    m_sizeInBytes = sizeInBytes;
    m_resultSize = resultSize;

    cudaStatus = hipMallocAsync((void**)&d_data, m_sizeInBytes, useStream);
    assert(cudaStatus == hipSuccess && "hipMalloc failed!");

    cudaStatus = hipMallocAsync((void**)&d_result, m_resultSize, useStream);
    assert(cudaStatus == hipSuccess && "hipMalloc failed!");
}

__host__
void Cuda::deallocate() {
    cudaStatus = hipFree(d_data);// it was said to -> hipFree ( void* devPtr )Frees memory on the device.
    assert(cudaStatus == hipSuccess && "not able to deallocate d_data");
    cudaStatus = hipFree(d_result);
    assert(cudaStatus == hipSuccess && "not able to deallocate d_result");
}

__host__
void Cuda::uploadToDevice(hipStream_t providedStream, const uint8_t* data) {
    hipStream_t useStream = providedStream == nullptr ? stream : providedStream;
    // If m_sizeInBytes is 0 we have not allocated enought memory.
    assert(m_sizeInBytes != 0);
    cudaStatus = hipMemcpyAsync(d_data, data, m_sizeInBytes, hipMemcpyHostToDevice, useStream);
    assert(cudaStatus == hipSuccess && "not able to trainsfer data, between host and device");
}

__host__ 
void Cuda::download(hipStream_t providedStream, uint8_t*& h_Data) {
    // If m_resultSize is 0, we have not allocated enought memory 
    hipStream_t useStream = providedStream == nullptr ? stream : providedStream;
    assert(m_resultSize != 0);
    cudaStatus = hipMemcpyAsync(h_Data, d_result, m_resultSize, hipMemcpyDeviceToHost, useStream);
    assert(cudaStatus == hipSuccess && "not able to transfer device to host!");
}

__host__
void Cuda::sync(hipStream_t providedStream) {
    hipStream_t useStream = providedStream == nullptr ? stream : providedStream;
    cudaStatus = hipStreamSynchronize(useStream);
    assert(cudaStatus == hipSuccess && "not able to sync!");
}

void Cuda::debugOutPutFile(uint8_t*& h_cpy) {
    FILE* fileWr;
    fileWr = fopen("writingFile.ppm", "w+");
    fprintf(fileWr, "%s %d %d %d ", "P6", params.width , params.height, 255);
    fclose(fileWr);

    fileWr = fopen("writingFile.ppm", "ab+");
    fwrite(reinterpret_cast<const char*>(&h_cpy[0]), 1, params.size(), fileWr);
    fclose(fileWr);
    fileWr = nullptr;
}

__host__
Cuda::~Cuda() {
    deallocate();
}